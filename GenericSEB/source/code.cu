#include "hip/hip_runtime.h"
#include "Constants.h"

extern "C"{

    #define K1 607.76f
    #define Tao_24h 0.59238046f
    #define K2 1260.56f
    #define UR 74.01f
    #define Kt 1.0f
    #define reflectanciaAtmosfera 0.03f
    #define width 1000.0f
    #define StefanBoltzman 5.67E-8f
    #define L 0.1f
    #define julianDay 85.0f
    #define Uref 1.63f
    #define Ta 31.03f
    #define height 1000.0f
    #define P 99.9f
    #define Rg_24h 243.77081f
    #define S 1367.0f
    #define latitude -16.56f
    #define Z 53.178f

    #define dr 1.0035436f
    #define cosZ 0.8005013f
    #define declinacaoSolar 0.03518365f
    #define anguloHorarioNascerSol 1.5603299f
    #define rad_solar_toa 35.540268f
    #define Rg_24h_mj 21.061798f
    #define transmissividade24h 0.59261787f
    #define ea 3.3301563f
    #define W 48.675568f
    #define transmissividade 0.7045728f
    #define emissivityAtm 0.8551312f
    #define SWd 739.1485f
    #define LWdAtm 415.08633f

    __constant__ float parameterAlbedo[] = {0.293f,0.274f,0.233f,0.157f,0.033f,0.0f,0.011f};

    __constant__ float calibration1[] = {-1.52f,193.0f,1957.0f};
    __constant__ float calibration2[] = {-2.84f,365.0f,1826.0f};
    __constant__ float calibration3[] = {-1.17f,264.0f,1554.0f};
    __constant__ float calibration4[] = {-1.51f,221.0f,1036.0f};
    __constant__ float calibration5[] = {-0.37f,30.2f,215.0f};
    __constant__ float calibration6[] = {1.2378f,15.303f,1.0f};
    __constant__ float calibration7[] = {-0.15f,16.5f,80.67f};

    __device__ void execute_sub(
        float pixel1,
        float pixel2,
        float pixel3,
        float pixel4,
        float pixel5,
        float pixel6,
        float pixel7,

        float * maxTsVet,
        float * minIndexVet,
        float * minTsVet,
        float * maxIndexVet,
        float * rnHotVet,
        float * gHotVet,
        float * saviHotVet
    ){

        float albedo = 0.0f;
        float NDVI = 0.0f;
        float SAVI = 0.0f;
        float IAF = 0.0f;
        float emissividadeNB = 0.0f;
        float emissivity = 0.0f;
        float Ts = 0.0f;
        float LWd = 0.0f;
        float Rn = 0.0f;
        float G0 = 0.0f;
        float sebta = 0.0f;
        float sumBandas = 0.0f;
        float banda1=calibration1[0]+((calibration1[1]-calibration1[0])/255.0f)*pixel1;

        float banda2=calibration2[0]+((calibration2[1]-calibration2[0])/255.0f)*pixel2;

        float banda3=calibration3[0]+((calibration3[1]-calibration3[0])/255.0f)*pixel3;

        float banda4=calibration4[0]+((calibration4[1]-calibration4[0])/255.0f)*pixel4;

        float banda5=calibration5[0]+((calibration5[1]-calibration5[0])/255.0f)*pixel5;

        float banda6=calibration6[0]+((calibration6[1]-calibration6[0])/255.0f)*pixel6;

        float banda7=calibration7[0]+((calibration7[1]-calibration7[0])/255.0f)*pixel7;

        sumBandas = 0.0f;
        float bandaRefletida1=(pi*banda1)/(calibration1[2]*cosZ*dr);

        sumBandas += parameterAlbedo[0]*bandaRefletida1;
        float bandaRefletida2=(pi*banda2)/(calibration2[2]*cosZ*dr);

        sumBandas += parameterAlbedo[1]*bandaRefletida2;
        float bandaRefletida3=(pi*banda3)/(calibration3[2]*cosZ*dr);

        sumBandas += parameterAlbedo[2]*bandaRefletida3;
        float bandaRefletida4=(pi*banda4)/(calibration4[2]*cosZ*dr);

        sumBandas += parameterAlbedo[3]*bandaRefletida4;
        float bandaRefletida5=(pi*banda5)/(calibration5[2]*cosZ*dr);

        sumBandas += parameterAlbedo[4]*bandaRefletida5;
        float bandaRefletida6=(pi*banda6)/(calibration6[2]*cosZ*dr);

        sumBandas += parameterAlbedo[5]*bandaRefletida6;
        float bandaRefletida7=(pi*banda7)/(calibration7[2]*cosZ*dr);

        sumBandas += parameterAlbedo[6]*bandaRefletida7;
            albedo = (sumBandas-reflectanciaAtmosfera)/(transmissividade*transmissividade);

            NDVI = (bandaRefletida4-bandaRefletida3)/(bandaRefletida4+bandaRefletida3);

            SAVI = ((1.0f+L)*(bandaRefletida4-bandaRefletida3))/(L+bandaRefletida4+bandaRefletida3);

            IAF = (-log((0.69f-SAVI)/0.59f)/0.91f);

            if(SAVI <= 0.1f ){
                IAF = 0.0f;

            }

            if(SAVI >= 0.687f ){
                IAF = 6.0f;

            }

            emissividadeNB = 0.97f+0.0033f*IAF;

            if(IAF >= 3.0f ){
                emissividadeNB = 0.98f;

            }

            if(NDVI <= 0.0f ){
                emissividadeNB = 0.99f;

            }

            emissivity = 0.95f+0.01f*IAF;

            if(IAF >= 3.0f ){
                emissivity = 0.98f;

            }

            if(NDVI <= 0.0f ){
                emissivity = 0.985f;

            }

            Ts = K2/log(((emissividadeNB*K1)/banda6)+1.0f);

            LWd = emissivity*StefanBoltzman*(pow(Ts,4.0f));

            Rn = ((1.0f-albedo)*SWd)+(emissivity*(LWdAtm)-LWd);

            G0 = Rn*(((Ts-T0)/albedo)*(0.0038f*albedo+0.0074f*albedo*albedo)*(1.0f-0.98f*pow(NDVI,4.0f)));

            sebta = (0.5f)*((2.0f*bandaRefletida4+1.0f)-sqrt((pow((2.0f*bandaRefletida4+1.0f),2.0f)-8.0f*(bandaRefletida4-bandaRefletida3))));

            if(sebta <= *minIndexVet){
                if(Ts >= *maxTsVet){
                    *maxTsVet=Ts;
                    *minIndexVet=sebta;
                    *rnHotVet=Rn;
                    *gHotVet=G0;
                    *saviHotVet=SAVI;
                }
            }
            if(sebta >= *maxIndexVet){
                if(Ts <= *minTsVet){
                    *minTsVet=Ts;
                    *maxIndexVet=sebta;
                }
            }

    }

    __global__ void execute(
        float * pixel1,
        float * pixel2,
        float * pixel3,
        float * pixel4,
        float * pixel5,
        float * pixel6,
        float * pixel7,

        float * maxTsVet,
        float * minIndexVet,
        float * minTsVet,
        float * maxIndexVet,
        float * rnHotVet,
        float * gHotVet,
        float * saviHotVet,
        int * parameters){
        int size = 10000;
        int idx = blockIdx.x*blockDim.x + threadIdx.x;
        int ind = idx;
        if(idx < size){
            maxTsVet[ind]=-99999.0f;
            minIndexVet[ind]=99999.0f;
            minTsVet[ind]=99999.0f;
            maxIndexVet[ind]=-99999.0f;
            rnHotVet[ind]=-99999.0f;
            gHotVet[ind]=-99999.0f;
            saviHotVet[ind]=-99999.0f;
            for(int i=0;i<parameters[2];i++){
                idx = ind*parameters[2]+i;
            if(idx < 2 && !(pixel1[idx] == pixel2[idx] && pixel1[idx] == pixel3[idx] && pixel1[idx] == pixel4[idx] && pixel1[idx] == pixel5[idx] && pixel1[idx] == pixel6[idx] && pixel1[idx] == pixel7[idx])){
                execute_sub(
                    pixel1[idx],
                    pixel2[idx],
                    pixel3[idx],
                    pixel4[idx],
                    pixel5[idx],
                    pixel6[idx],
                    pixel7[idx],
                    (maxTsVet+ind),
                    (minIndexVet+ind),
                    (minTsVet+ind),
                    (maxIndexVet+ind),
                    (rnHotVet+ind),
                    (gHotVet+ind),
                    (saviHotVet+ind)
                );
            }
            }
        }
    }
}


