#include "hip/hip_runtime.h"
#include "Constantes.h"

extern "C"{

 __device__ void SEBAL_EnergyBalance_G(
			float SWd,
			float LWd,
			float albedo,
			float emissivity,
			float LST_K,
			float NDVI,
			float Uref,
			float SAVI,
			float a,
			float b,
			float Rg_24h,
			float Tao_24h,

			float * z0m,
			float * U_star,
			float * r_ah,
			float * Rn,
			float * G0,
			float * H,
			float * LE,
			float * evap_fr,
			float * Rn_24h,
			float * LE_24h,
			float * ET_24h)
{

	*z0m = expf(-5.809f+5.62f*SAVI);

	/* Classification */
	bool I_snow = (NDVI<0.0f) && (albedo>0.47f);
	bool I_water = (NDVI==-1.0f);
	
/*	% NOTE: esat_WL is only used for the wet-limit. To get a true upperlimit for the sensible heat
	% the Landsurface Temperature is used as a proxy instead of air temperature.
	%% Net Radiation */
	float SWnet = (1.0f - albedo) * SWd; /* Shortwave Net Radiation [W/m2] */
	float LWnet = emissivity*LWd - emissivity*Sigma_SB*LST_K*LST_K*LST_K*LST_K; /* Longwave Net Radiation [W/m2] */
	*Rn = SWnet+LWnet; /* Total Net Radiation [W/m2] */
	
	/* Ground Heat Flux */
	/* Kustas et al 1993 */
	/* Kustas, W.P., Daughtry, C.S.T. van Oevelen P.J., 
	Analatytical Treatment of Relationships between Soil heat flux/net radiation and Vegetation Indices, 
	Remote sensing of environment,46:319-330 (1993) */
	*G0 = *Rn * (((LST_K-T0)/albedo)*(0.0038f*albedo+0.0074*albedo*albedo)*(1.0f-0.98f*NDVI*NDVI*NDVI*NDVI)); 

	if (I_water || I_snow)
	{
		*G0= 0.3f* *Rn; 
	}
	
	*U_star = k*Uref/logf(z200/ *z0m);
	
	*r_ah = logf(z2/z1)/(*U_star*k);
	
	*H = p*cp*(b+a*(LST_K - T0))/ *r_ah;

	*LE = *Rn - *H	- *G0;
	
	/* Evaporative fraction */
	*evap_fr = 0.0f;
	if ((*Rn - *G0) != 0.0f)
	{
		*evap_fr = *LE/(*Rn-*G0); /* evaporative fraction [] */
	}
	else
	{
		*evap_fr = 1.0f; /* evaporative fraction upper limit [] (for negative available energy) */
	}

	*Rn_24h = Rg_24h*(1-albedo) - 110.0f*Tao_24h;
	*LE_24h = *evap_fr * *Rn_24h;

	*ET_24h = (*evap_fr * *Rn_24h*86.4f)/2450.0f;
}


 __global__ void SEBAL_EnergyBalance_Kernel(
			int * comptMask,
			float * SWd,
			float * LWd,
			float * albedo,
			float * emissivity,
			float * LST_K,
			float * NDVI,
			float * Uref,
			float * SAVI,
			float * a,
			float * b,
			float * Rg_24h,
			float * Tao_24h,

			float * z0m,
			float * Ustar,
			float * r_ah,
			float * Rn,
			float * G0,
			float * H,
			float * LE,
			float * evap_fr,
			float * Rn_24h,
			float * LE_24h,
			float * ET_24h,
			int  DataSize)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx<DataSize){
		*(z0m+idx)=-9999.0f;
		*(Ustar+idx)=-9999.0f;
		*(r_ah+idx)=-9999.0f;
		*(Rn+idx)=-9999.0f;
		*(G0+idx)=-9999.0f;
		*(H+idx)=-9999.0f;
		*(LE+idx)=-9999.0f;
		*(evap_fr+idx)=-9999.0f;
		*(Rn_24h+idx)=-9999.0f;
		*(LE_24h+idx)=-9999.0f;
		*(ET_24h+idx)=-9999.0f;

		if(comptMask[idx] == 1){
			SEBAL_EnergyBalance_G(
				SWd[idx],
				LWd[idx],
				albedo[idx],
				emissivity[idx],
				LST_K[idx],
				NDVI[idx],
				Uref[idx],
				SAVI[idx],
				a[0],
				b[0],
				Rg_24h[idx],
				Tao_24h[idx],
				(z0m+idx),
				(Ustar+idx),
				(r_ah+idx),
				(Rn+idx),
				(G0+idx),
				(H+idx),
				(LE+idx),
				(evap_fr+idx),
				(Rn_24h+idx),
				(LE_24h+idx),
				(ET_24h+idx));
		}
	}
}

}
