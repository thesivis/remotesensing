#include "hip/hip_runtime.h"
#include "Constants.h"

    __device__ float X(float Zref_m, float L) {
        return (float) (sqrtf(sqrtf((1.0f - 16.0f * Zref_m / L))));
    }

    __device__ float Psim(float L) {
        if (L < 0.0f) {
            /* unstable */
            float x200 = X(200, L);
            return (float) (2.0f * logf((1.0f + x200) / 2.0f) + logf((1.0f + x200 * x200) / (2.0f)) - 2.0f * atanf(x200) + 0.5f * pi);
        } else if (L > 0.0f) {
            /* stable */
            return (-5 * (2 / L));
        } else {
            return (0);
        }
    }

    __device__ float Psih(float Zref_h, float L) {
        if (L < 0.0f) {
            /* unstable */
            float x = X(Zref_h, L);
            return (float) (2.0f * logf((1.0f + x * x) / 2.0f));
        } else if (L > 0.0f) {
            /* stable */
            return (-5 * (2 / L));
        } else {
            return (0);
        }
    }

extern "C"{

    #define K1 607.76f
    #define K2 1260.56f
    #define Kt 1.0f
    #define L 0.1f
    #define Uref 2.24f
    #define indexMin 203.69563f
    #define height 6797.0f
    #define P 99.9f
    #define Rg_24h 204.60323f
    #define S 1367.0f
    #define RnHot 499.16528f
    #define h_veg 28.0f
    #define GHot 89.82469f
    #define Z 50.4362f
    #define UR 78.79f
    #define Tao_24h 0.53428197f
    #define U_z 32.0f
    #define reflectanciaAtmosfera 0.03f
    #define StefanBoltzman 5.67E-8f
    #define width 7989.0f
    #define julianDay 101.0f
    #define Ta 29.2f
    #define SAVI_hot 0.097473025f
    #define latitude -16.56f
    #define indexMax 311.61884f

    #define dr 0.9944873f
    #define cosZ 0.7709158f
    #define declinacaoSolar 0.14475246f
    #define anguloHorarioNascerSol 1.527437f
    #define rad_solar_toa 33.07308f
    #define Rg_24h_mj 17.67772f
    #define transmissividade24h 0.5345048f
    #define ea 3.1922805f
    #define W 46.747234f
    #define transmissividade 0.702253f
    #define emissivityAtm 0.8510801f
    #define SWd 708.1516f
    #define LWdAtm 403.26764f

    __constant__ float parameterAlbedo[] = {0.2934178f,0.27377668f,0.23299503f,0.15533003f,0.032235477f,0.0f,0.012095051f};

    __constant__ float calibration1[] = {-1.52f,193.0f,1957.0f};
    __constant__ float calibration2[] = {-2.84f,365.0f,1826.0f};
    __constant__ float calibration3[] = {-1.17f,264.0f,1554.0f};
    __constant__ float calibration4[] = {-1.51f,221.0f,1036.0f};
    __constant__ float calibration5[] = {-0.37f,30.2f,215.0f};
    __constant__ float calibration6[] = {1.2378f,15.303f,1.0f};
    __constant__ float calibration7[] = {-0.15f,16.5f,80.67f};

    __device__ void execute_sub(
        short pixel1,
        short pixel2,
        short pixel3,
        short pixel4,
        short pixel5,
        short pixel6,
        short pixel7,

        float * ET_24h    ){

        float z0m = 0.0f;
        float U_star = 0.0f;
        float H = 0.0f;
        float r_ah = 0.0f;
        float albedo = 0.0f;
        float NDVI = 0.0f;
        float SAVI = 0.0f;
        float IAF = 0.0f;
        float emissividadeNB = 0.0f;
        float emissivity = 0.0f;
        float Ts = 0.0f;
        float LWd = 0.0f;
        float Rn = 0.0f;
        float G0 = 0.0f;
        float LE = 0.0f;
        float evap_fr = 0.0f;
        float Rn_24h = 0.0f;
        float LE_24h = 0.0f;
            z0m = 0.0f;

            U_star = 0.0f;

            H = 0.0f;

            r_ah = 0.0f;

        float sumBandas = 0.0f;
        float banda1=calibration1[0]+((calibration1[1]-calibration1[0])/255.0f)*pixel1;

        float banda2=calibration2[0]+((calibration2[1]-calibration2[0])/255.0f)*pixel2;

        float banda3=calibration3[0]+((calibration3[1]-calibration3[0])/255.0f)*pixel3;

        float banda4=calibration4[0]+((calibration4[1]-calibration4[0])/255.0f)*pixel4;

        float banda5=calibration5[0]+((calibration5[1]-calibration5[0])/255.0f)*pixel5;

        float banda6=calibration6[0]+((calibration6[1]-calibration6[0])/255.0f)*pixel6;

        float banda7=calibration7[0]+((calibration7[1]-calibration7[0])/255.0f)*pixel7;

        sumBandas = 0.0f;
        float bandaRefletida1=(pi*banda1)/(calibration1[2]*cosZ*dr);

        sumBandas += parameterAlbedo[0]*bandaRefletida1;
        float bandaRefletida2=(pi*banda2)/(calibration2[2]*cosZ*dr);

        sumBandas += parameterAlbedo[1]*bandaRefletida2;
        float bandaRefletida3=(pi*banda3)/(calibration3[2]*cosZ*dr);

        sumBandas += parameterAlbedo[2]*bandaRefletida3;
        float bandaRefletida4=(pi*banda4)/(calibration4[2]*cosZ*dr);

        sumBandas += parameterAlbedo[3]*bandaRefletida4;
        float bandaRefletida5=(pi*banda5)/(calibration5[2]*cosZ*dr);

        sumBandas += parameterAlbedo[4]*bandaRefletida5;
        float bandaRefletida6=(pi*banda6)/(calibration6[2]*cosZ*dr);

        sumBandas += parameterAlbedo[5]*bandaRefletida6;
        float bandaRefletida7=(pi*banda7)/(calibration7[2]*cosZ*dr);

        sumBandas += parameterAlbedo[6]*bandaRefletida7;
            albedo = (sumBandas-reflectanciaAtmosfera)/(transmissividade*transmissividade);

            NDVI = (bandaRefletida4-bandaRefletida3)/(bandaRefletida4+bandaRefletida3);

            SAVI = ((1.0f+L)*(bandaRefletida4-bandaRefletida3))/(L+bandaRefletida4+bandaRefletida3);

            IAF = (-log((0.69f-SAVI)/0.59f)/0.91f);

            if(SAVI <= 0.1f ){
                IAF = 0.0f;

            }

            if(SAVI >= 0.687f ){
                IAF = 6.0f;

            }

            emissividadeNB = 0.97f+0.0033f*IAF;

            if(IAF >= 3.0f ){
                emissividadeNB = 0.98f;

            }

            if(NDVI <= 0.0f ){
                emissividadeNB = 0.99f;

            }

            emissivity = 0.95f+0.01f*IAF;

            if(IAF >= 3.0f ){
                emissivity = 0.98f;

            }

            if(NDVI <= 0.0f ){
                emissivity = 0.985f;

            }

            Ts = K2/log(((emissividadeNB*K1)/banda6)+1.0f);

                float constZ=0.12f;
                float z0=constZ*h_veg;
                
                float Ustar = (float) (Uref*k/logf(U_z/z0));
                float U200 = (float) (Ustar*logf(z200 / z0)/k);
                z0m = (float) expf(-5.809f + 5.62f * SAVI);
                U_star = (float) (k * U200 / logf(z200 / z0m));
                r_ah = (float) (logf(z2 / z1) / (U_star * k));

                float LHot = 0.0f;
                float tm_200Hot = 0.0f;
                float th_2Hot = 0.0f;
                float th_0_1Hot = 0.0f;

                float LPixel = 0.0f;
                float tm_200Pixel = 0.0f;
                float th_2Pixel = 0.0f;
                float th_0_1Pixel = 0.0f;

                float HHot = RnHot - GHot;
                float a = 0.0f;
                float b = 0.0f;
                float errorH = 10.0f;
                float r_ah_anteriorHot = 0.0f;
                int step = 1;
                float z0mHot = (float) expf(-5.809f + 5.62f * SAVI_hot);
                float U_starHot = (float) (k * U200 / logf(z200 / z0mHot));
                float r_ahHot = (float) (logf(z2 / z1) / (U_starHot * k));
                while (errorH > MaxAllowedError && step < 15) {

                    a = ((HHot) * r_ahHot) / (p * cp * (indexMax - indexMin));
                    b = -a * (indexMin - T0);

                    //PARTE DO PIXEL QUENTE
                    HHot = p * cp * (b + a * (indexMax - T0)) / r_ahHot;
                    LHot = (float) (-(p * cp * U_starHot * U_starHot * U_starHot * (indexMax)) / (k * g * HHot));

                    tm_200Hot = Psim(LHot);
                    th_2Hot = Psih(z2, LHot);
                    th_0_1Hot = Psih(z1, LHot);

                    U_starHot = (float) (k * U200 / (logf(z200 / z0mHot) - tm_200Hot));
                    r_ah_anteriorHot = r_ahHot;
                    r_ahHot = (float) ((logf(z2 / z1) - th_2Hot + th_0_1Hot) / (U_starHot * k));

                    //PARTE DE CADA PIXEL
                    H = p * cp * (b + a * (Ts - T0)) / r_ah;
                    LPixel = (float) (-(p * cp * U_star * U_star * U_star * (Ts)) / (k * g * H));

                    tm_200Pixel = Psim(LPixel);
                    th_2Pixel = Psih(z2, LPixel);
                    th_0_1Pixel = Psih(z1, LPixel);

                    U_star = (float) (k * U200 / (logf(z200 / z0m) - tm_200Pixel));
                    r_ah = (float) ((logf(z2 / z1) - th_2Pixel + th_0_1Pixel) / (U_star * k));

                    errorH = fabsf(((r_ahHot - r_ah_anteriorHot) * 100) / r_ahHot);

                    step++;
                }

            LWd = emissivity*StefanBoltzman*(pow(Ts,4.0f));

            Rn = ((1.0f-albedo)*SWd)+(emissivity*(LWdAtm)-LWd);

            G0 = Rn*(((Ts-T0)/albedo)*(0.0038f*albedo+0.0074f*albedo*albedo)*(1.0f-0.98f*pow(NDVI,4.0f)));

            LE = Rn-H-G0;

            evap_fr = LE/(Rn-G0);

            Rn_24h = Rg_24h*(1.0f-albedo)-110.0f*Tao_24h;

            LE_24h = evap_fr*Rn_24h;

            *ET_24h = (evap_fr*Rn_24h*86.4f)/2450.0f;

    }

    __global__ void execute(
        short * pixel1,
        short * pixel2,
        short * pixel3,
        short * pixel4,
        short * pixel5,
        short * pixel6,
        short * pixel7,

        float * ET_24h,
        int * parameters){
        int size = 54301233;
        int idx = blockIdx.x*blockDim.x + threadIdx.x;
        int ind = idx;
        if(idx < size){
            if(idx < 54301233 && !(pixel1[idx] == pixel2[idx] && pixel1[idx] == pixel3[idx] && pixel1[idx] == pixel4[idx] && pixel1[idx] == pixel5[idx] && pixel1[idx] == pixel6[idx] && pixel1[idx] == pixel7[idx])){
                execute_sub(
                    pixel1[idx],
                    pixel2[idx],
                    pixel3[idx],
                    pixel4[idx],
                    pixel5[idx],
                    pixel6[idx],
                    pixel7[idx],
                    (ET_24h+idx)                );
            }
        }
    }
}


