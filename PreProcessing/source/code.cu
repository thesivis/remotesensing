#include "hip/hip_runtime.h"
#include "Constants.h"

extern "C"{

    #define StefanBoltzman 5.67E-8f
    #define K1 607.76f
    #define K2 1260.56f
    #define latitude -16.56f
    #define Kt 1.0f
    #define L 0.1f
    #define Ta 20.53f
    #define TC 204.02965f
    #define julianDay 157.0f
    #define P 99.3f
    #define Tao_24h 0.59930485f
    #define S 1367.0f
    #define TH 297.83206f
    #define Uref 2.24f
    #define width 7947.0f
    #define Rg_24h 181.61319f
    #define Z 39.8911f
    #define reflectanciaAtmosfera 0.03f
    #define UR 68.59f
    #define height 6799.0f

    #define dr 0.9701286f
    #define cosZ 0.6413305f
    #define declinacaoSolar 0.3970275f
    #define anguloHorarioNascerSol 1.4457928f
    #define rad_solar_toa 26.170544f
    #define Rg_24h_mj 15.69138f
    #define transmissividade24h 0.5995817f
    #define ea 1.657021f
    #define W 25.135906f
    #define transmissividade 0.7112397f
    #define emissivityAtm 0.7840079f
    #define SWd 591.093f
    #define LWdAtm 330.6747f

    __constant__ float parameterAlbedo[] = {0.293f,0.274f,0.233f,0.157f,0.033f,0.0f,0.011f};

    __constant__ float calibration1[] = {-1.52f,193.0f,1957.0f};
    __constant__ float calibration2[] = {-2.84f,365.0f,1826.0f};
    __constant__ float calibration3[] = {-1.17f,264.0f,1554.0f};
    __constant__ float calibration4[] = {-1.51f,221.0f,1036.0f};
    __constant__ float calibration5[] = {-0.37f,30.2f,215.0f};
    __constant__ float calibration6[] = {1.2378f,15.303f,1.0f};
    __constant__ float calibration7[] = {-0.15f,16.5f,80.67f};

    __device__ void execute_sub(
        short pixel1,
        short pixel2,
        short pixel3,
        short pixel4,
        short pixel5,
        short pixel6,
        short pixel7,

        float * ET_24h
    ){

        float albedo = 0.0f;
        float NDVI = 0.0f;
        float SAVI = 0.0f;
        float mSAVI = 0.0f;
        float IAF = 0.0f;
        float emissividadeNB = 0.0f;
        float emissivity = 0.0f;
        float Ts = 0.0f;
        float LWd = 0.0f;
        float Rn = 0.0f;
        float G0 = 0.0f;
        float evap_fr = 0.0f;
        float H = 0.0f;
        float LE = 0.0f;
        float Rn_24h = 0.0f;
        float LE_24h = 0.0f;
        float sumBandas = 0.0f;
        float banda1=calibration1[0]+((calibration1[1]-calibration1[0])/255.0f)*pixel1;

        float banda2=calibration2[0]+((calibration2[1]-calibration2[0])/255.0f)*pixel2;

        float banda3=calibration3[0]+((calibration3[1]-calibration3[0])/255.0f)*pixel3;

        float banda4=calibration4[0]+((calibration4[1]-calibration4[0])/255.0f)*pixel4;

        float banda5=calibration5[0]+((calibration5[1]-calibration5[0])/255.0f)*pixel5;

        float banda6=calibration6[0]+((calibration6[1]-calibration6[0])/255.0f)*pixel6;

        float banda7=calibration7[0]+((calibration7[1]-calibration7[0])/255.0f)*pixel7;

        sumBandas = 0.0f;
        float bandaRefletida1=(pi*banda1)/(calibration1[2]*cosZ*dr);

        sumBandas += parameterAlbedo[0]*bandaRefletida1;
        float bandaRefletida2=(pi*banda2)/(calibration2[2]*cosZ*dr);

        sumBandas += parameterAlbedo[1]*bandaRefletida2;
        float bandaRefletida3=(pi*banda3)/(calibration3[2]*cosZ*dr);

        sumBandas += parameterAlbedo[2]*bandaRefletida3;
        float bandaRefletida4=(pi*banda4)/(calibration4[2]*cosZ*dr);

        sumBandas += parameterAlbedo[3]*bandaRefletida4;
        float bandaRefletida5=(pi*banda5)/(calibration5[2]*cosZ*dr);

        sumBandas += parameterAlbedo[4]*bandaRefletida5;
        float bandaRefletida6=(pi*banda6)/(calibration6[2]*cosZ*dr);

        sumBandas += parameterAlbedo[5]*bandaRefletida6;
        float bandaRefletida7=(pi*banda7)/(calibration7[2]*cosZ*dr);

        sumBandas += parameterAlbedo[6]*bandaRefletida7;
            albedo = (sumBandas-reflectanciaAtmosfera)/(transmissividade*transmissividade);

            NDVI = (bandaRefletida4-bandaRefletida3)/(bandaRefletida4+bandaRefletida3);

            SAVI = ((1.0f+L)*(bandaRefletida4-bandaRefletida3))/(L+bandaRefletida4+bandaRefletida3);

            mSAVI = (0.5f)*((2.0f*bandaRefletida4+1.0f)-sqrt((pow((2.0f*bandaRefletida4+1.0f),2.0f)-8.0f*(bandaRefletida4-bandaRefletida3))));

            IAF = (-log((0.69f-SAVI)/0.59f)/0.91f);

            if(SAVI <= 0.1f ){
                IAF = 0.0f;

            }

            if(SAVI >= 0.687f ){
                IAF = 6.0f;

            }

            emissividadeNB = 0.97f+0.0033f*IAF;

            if(IAF >= 3.0f ){
                emissividadeNB = 0.98f;

            }

            if(NDVI <= 0.0f ){
                emissividadeNB = 0.99f;

            }

            emissivity = 0.95f+0.01f*IAF;

            if(IAF >= 3.0f ){
                emissivity = 0.98f;

            }

            if(NDVI <= 0.0f ){
                emissivity = 0.985f;

            }

            Ts = K2/log(((emissividadeNB*K1)/banda6)+1.0f);

            LWd = emissivity*StefanBoltzman*(pow(Ts,4.0f));

            Rn = ((1.0f-albedo)*SWd)+(emissivity*(LWdAtm)-LWd);

            G0 = Rn*(((Ts-T0)/albedo)*(0.0038f*albedo+0.0074f*albedo*albedo)*(1.0f-0.98f*NDVI*NDVI*NDVI*NDVI));

            evap_fr = (TH-Ts)/(TH-TC);

            H = (1.0f-evap_fr)*(Rn-G0);

            LE = (evap_fr)*(Rn-G0);

            Rn_24h = Rg_24h*(1.0f-albedo)-110.0f*Tao_24h;

            LE_24h = evap_fr*Rn_24h;

            *ET_24h = (evap_fr*Rn_24h*86.4f)/2450.0f;

    }

    __global__ void execute(
        short * pixel1,
        short * pixel2,
        short * pixel3,
        short * pixel4,
        short * pixel5,
        short * pixel6,
        short * pixel7,

        float * ET_24h,
        int * parameters){
        int size = parameters[0];
        int idx = blockIdx.x*blockDim.x + threadIdx.x;
        int ind = idx;
        if(idx < size){
            if(idx < parameters[0] && !(pixel1[idx] == pixel2[idx] && pixel1[idx] == pixel3[idx] && pixel1[idx] == pixel4[idx] && pixel1[idx] == pixel5[idx] && pixel1[idx] == pixel6[idx] && pixel1[idx] == pixel7[idx])){
                execute_sub(
                    pixel1[idx],
                    pixel2[idx],
                    pixel3[idx],
                    pixel4[idx],
                    pixel5[idx],
                    pixel6[idx],
                    pixel7[idx],
                    (ET_24h+idx)
                );
            }
        }
    }
}

